#include <hip/hip_runtime.h>
#define N_STATES (N_THREADS * N_BLOCKS)

int main() {
    int num_streams = 4; // Number of CUDA streams. This should be set via a makefile configiuration.
    

    // Initialize CUDA
    hipSetDevice(0);
    hiprandState *dev_states;

    hipMalloc((void **)&dev_states, N_STATES * sizeof(hiprandState));

    init_rng<<<N_BLOCKS, N_THREADS>>>(dev_states, time(NULL), N_STATES);


    // Create CUDA streams
    hipStream_t stream[num_streams];
    for (int i = 0; i < num_streams; i++) {
        hipStreamCreate(&stream[i]);
    }

    // Allocate memory on the CUDA device
    float *d_data[num_streams];
    for (int i = 0; i < num_streams; i++) {
        hipMalloc(&d_data[i], stream_size * sizeof(float));
    }

    // Launch CUDA kernels on multiple streams
    for (int i = 0; i < num_streams; i++) {
        hipMemcpyAsync(&d_data[i][0], &stream_size, sizeof(float), hipMemcpyHostToDevice, stream[i]);
        switch(model_in) {
            case 1:
                launchModel1(stream[i], dev_states, launch_struct_ptr);
                break;
            case 2:
                launchModel2(stream[i], dev_states, launch_struct_ptr);
                break;
            case 3:
                launchModel3(stream[i], dev_states, launch_struct_ptr);
                break;
            default:
                printf("Invalid model selection.\n");
                break;
        }
        hipMemcpyAsync(&h_data[i][0], &d_data[i][0], stream_size * sizeof(float), hipMemcpyDeviceToHost, stream[i]);
    }

    // Wait for CUDA streams to finish
    for (int i = 0; i < num_streams; i++) {
        hipStreamSynchronize(stream[i]);
    }

    // Destroy CUDA streams
    for (int i = 0; i < num_streams; i++) {
        hipStreamDestroy(stream[i]);
    }

    // Free allocated memory on the CUDA device
    for (int i = 0; i < num_streams; i++) {
        hipFree(d_data[i]);
    }

    // Free the RNG on the device
    hipFree(dev_states);

    return 0;
}
