#include "../include/model_wrappers.h"

//Todo this should go to helpers
void init_model(ising_model_config* launch_struct) {
    // Add model specific launch parameters
    switch(launch_struct->model_id) {
        case 1:
            launch_struct->element_size = sizeof(int);
            break;
        default:
            fprintf(stderr, "Invalid model selection.\n");
            break;
    }
    return;
}

void launch_mc_sweep(hipStream_t stream, hiprandState *state, ising_model_config* launch_struct, int *host_array, int *device_array, int stream_ix) {
    /*
      * This launches the original model. Single thread per grid.
      *
      * Updates to this should not let the function block it should add tasks to the stream.
      *
      * Firstly transfer any initial grid configuration to the device.
      * Then launch the kernel.
      * Then transfer the result back to the host.
      * 
      * Parameters:
      *    stream: cuda stream to use
      *    state: hiprandState array to use
      *    launch_struct: struct containing launch parameters
    */

    // Allocate memory for device array
    //hipMalloc((void **)&device_array, launch_struct->element_size * launch_struct->size[0] * launch_struct->size[1]);

    switch(launch_struct->starting_config) {
        case 0:
            // Load Grid from file
            if (launch_struct->input_file != NULL) {
                // If we have initial grid(s) to load, load them, and transfer it to the device 
                load_grid(stream, launch_struct, device_array);
                gpuErrchk( hipPeekAtLastError() );
                gpuErrchk( hipDeviceSynchronize() );
            } 
            else {
                fprintf(stderr, "No initial grid to load.\n");
            }
            break;
        case 1:
            // Random
            init_rand_grids<<<launch_struct->num_blocks, launch_struct->num_concurrent, 0, stream>>>(state, launch_struct->size[0], launch_struct->size[1], launch_struct->num_concurrent, device_array);
            gpuErrchk( hipPeekAtLastError() );
            gpuErrchk( hipDeviceSynchronize() );
            break;
        case 2:
            // All up
            fprintf(stderr, "All up\n");
            init_ud_grids<<<launch_struct->num_blocks, launch_struct->num_concurrent, 0, stream>>>(launch_struct->size[0], launch_struct->size[1], launch_struct->num_concurrent, device_array, 1);
            gpuErrchk( hipPeekAtLastError() );
            gpuErrchk( hipDeviceSynchronize() );
            fprintf(stderr, "All are up\n");
            break;
        case 3:
            // All down
            init_ud_grids<<<launch_struct->num_blocks, launch_struct->num_concurrent, 0, stream>>>(launch_struct->size[0], launch_struct->size[1], launch_struct->num_concurrent, device_array, -1);
            gpuErrchk( hipPeekAtLastError() );
            gpuErrchk( hipDeviceSynchronize() );
            break;
        default:
            fprintf(stderr, "Invalid starting configuration.\n");
            break;
    }

    // Get initial grid from device
    gpuErrchk( hipMemcpy(host_array, device_array, launch_struct->mem_size, hipMemcpyDeviceToHost) );

    //TODO: Create d_Pacc and d_neighbour list here and refactor precomputations to be flexible
    // Allocate memory for d_Pacc and d_neighbour_list
    int prob_size = 10;
    launch_struct->prob_size = prob_size;

    // Allocate device memory for d_Pacc and d_neighbour_list (there is potential here to put this in a faster memory location?)
    float* d_Pacc;
    int* d_neighbour_list;
    hipMalloc(&d_Pacc, prob_size * sizeof(float));
    hipMalloc(&d_neighbour_list, launch_struct->size[0] * launch_struct->size[1] * 4 * sizeof(int));
    fprintf(stderr, "Allocated memory for d_Pacc and d_neighbour_list\n");

    // Precompute
    preComputeProbs(launch_struct, d_Pacc);
    preComputeNeighbours(launch_struct, d_neighbour_list);
    fprintf(stderr, "Precomputed probs and neighbours\n");

    // Allocate memory for magnetisation and energy
    float h_magnetisation[launch_struct->num_concurrent];
    float* d_magnetisation;
    hipMalloc(&d_magnetisation, launch_struct->num_concurrent * sizeof(float));
    float h_nucleation[launch_struct->num_concurrent];
    int* d_nucleation;
    hipMalloc(&d_nucleation, launch_struct->num_concurrent * sizeof(int));


    // Launch kernel
    for (int i = 0; i < launch_struct->iterations; i+=launch_struct->iter_per_step){
        mc_sweep<<<launch_struct->num_blocks, launch_struct->num_concurrent, 0, stream>>>(state, launch_struct->size[0], launch_struct->size[1], launch_struct->num_concurrent, device_array, launch_struct->inv_temperature, launch_struct->field, launch_struct->iter_per_step, d_neighbour_list, d_Pacc);
        gpuErrchk( hipPeekAtLastError() );
        gpuErrchk( hipStreamSynchronize(stream) );
        gpuErrchk( hipMemcpy(host_array, device_array, launch_struct->mem_size, hipMemcpyDeviceToHost));
        fprintf(stderr, "Iterations %d to %d\n", i, i+launch_struct->iter_per_step);
        gpuErrchk( hipPeekAtLastError() );
        gpuErrchk( hipDeviceSynchronize() );
        
        // Compute energy and magnetisation (GPU)
        compute_magnetisation<<<launch_struct->num_blocks, launch_struct->num_concurrent, 0, stream>>>(launch_struct->size[0], launch_struct->size[1], launch_struct->num_concurrent, launch_struct->nucleation_threshold, device_array, d_magnetisation, d_nucleation);
        gpuErrchk( hipPeekAtLastError() );
        gpuErrchk( hipStreamSynchronize(stream) );
        gpuErrchk( hipMemcpy(h_magnetisation, d_magnetisation, launch_struct->num_concurrent * sizeof(float), hipMemcpyDeviceToHost));
        gpuErrchk( hipPeekAtLastError() );
        // Write to file (CPU)
        outputGridToFile(launch_struct, host_array, h_magnetisation, i, stream_ix);
        // Check for full nucleation
        int full_nucleation = 0;
        for (int j = 0; j < launch_struct->num_concurrent; j++) {
            if (h_magnetisation[j] > launch_struct->nucleation_threshold) {
                fprintf(stderr, "Nucleation detected at iteration %d on grid %d\n", i, j+1);
                full_nucleation++;
            }
        }
        if (full_nucleation == launch_struct->num_concurrent) {
            fprintf(stderr, "Full nucleation on all grids \n");
            break;
        }
    }
    return;
}
