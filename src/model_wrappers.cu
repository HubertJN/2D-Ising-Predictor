#include "../include/model_wrappers.h"

int init_model(ising_model_config launch_struct) {
    // Add model specific launch parameters
    switch(launch_struct -> model_id) {
        case 1:
            launch_struct.element_size = 3*sizeof(int);
            break;
        case 2:
            launch_struct.element_size = 3*sizeof(int);
            break;
        default:
            fprintf(stderr, "Invalid model selection.\n");
            break;
    }
}


// 
int testModel1(hipStream_t stream, hiprandState *state, ising_model_config launch_struct, int *device_array) {
    // This tests the kernal that uses one thread to fill its grid sequentially.

    // Launch kernel
    test_1<<<launch_struct.num_blocks, launch_struct.num_concurrent, 0, stream>>>(state, device_array, launch_struct.size[0], launch_struct.size[1], launch_struct.num_concurrent);

    // Collect result
    int *array = (int *)malloc(launch_struct.size[0]*launch_struct.size[1] * launch_struct.element_size);
    hipMemcpy(array, device_array, launch_struct.element_size * launch_struct.size[0] * launch_struct.size[1] * launch_struct.num_concurrent, hipMemcpyDeviceToHost);
    
    // Each element is 3 ints so we multiply by 3 and add commas and newlines appropiately
    for(int i=0; i<launch_struct.size[0] * launch_struct.size[1] * launch_struct.num_concurrent * 3; i++) {
        if (i % 3 == 0) {
            fprintf(stdout, ", ");
        }
        if(i % (launch_struct.size[0] * 3) == 0) {
            fprintf(stdout, "\n");
            if (i % (launch_struct.size[0] * 3 * launch_struct.size[1]) == 0)
            {
                fprintf(stdout, "\n");
            }
        }
        fprintf(stdout, "%d", array[i]);
    }
    fprintf(stdout, "\n");
    return 0;
}

int testModel2(hipStream_t stream, hiprandState *state, ising_model_config launch_struct) {
    // This tests the kernal tht uses mutiple threads to fill its grid concurrently.
    int *device_array;
    // Allocate device memory
    hipMalloc((void **) &device_array, sizeof(float) * launch_struct.size[0] * launch_struct.size[1] * launch_struct.num_concurrent);

    // Launch kernel
    test_2<<<launch_struct.num_threads, launch_struct.num_concurrent, 0, stream>>>(state, device_array, launch_struct.num_threads, launch_struct.size[0], launch_struct.size[1], launch_struct.num_concurrent);

    // Collect result
    float *array;
    hipMemcpy((void**)&array, (void**)&device_array, sizeof(float) * launch_struct.size[0] * launch_struct.size[1] * launch_struct.num_concurrent, hipMemcpyDeviceToHost);

    // Print result (TODO: to file)
    for(int i=0; i<launch_struct.size[0] * launch_struct.size[1] * launch_struct.num_concurrent; i++) 
    {
        if(i % launch_struct.size[0] == 0) {
            fprintf(stdout, "\n");
            if (i % (launch_struct.size[0] * launch_struct.size[1]) == 0)
            {
                fprintf(stdout, "\n");
            }
        }
        fprintf(stdout, "%f ", array[i]);
    } 
   return 0; 
}


void launchModel1(hipStream_t stream, hiprandState *state, ising_model_config launch_struct) {
    // This model launches a kernel that is fully initilised on host working in shared memory. 
    // This function initilises the device memory and launches the kernel, then collects the result and frees the device memory.
    // The kernel is defined in kernels.h
    // The kernel is launched on the stream passed as argument.

    // Create pointers to device memory
    float *device_grid;
    // Allocate device memory
    hipMalloc((void **) &device_grid, sizeof(int) * launch_struct.size[0] * launch_struct.size[1] * launch_struct.num_concurrent);
    // Allocate pinned host memory
    int *grid;
    hipHostMalloc((void **) &grid, sizeof(int) * launch_struct.size[0] * launch_struct.size[1] * launch_struct.num_concurrent);
    int *magnetisation;
    hipHostMalloc((void **) &magnetisation, sizeof(int) * launch_struct.num_concurrent);

    for(int i=0; i<launch_struct.iterations; i+=launch_struct.iter_per_step) {
        // Launch kernel
        // ising_kernel_many<<<launch_struct.num_concurrent, 1, 0, stream>>>(state, device_grid, launch_struct.size[0], launch_struct.size[1], launch_struct.iter_per_step, launch_struct.temperature);
        // compute_magnetisation<<<launch_struct.num_concurrent, 1, 0, stream>>>(device_grid, launch_struct.size[0], launch_struct.size[1], magnetisation);
        
        // Collect result
        hipMemcpyAsync(grid, device_grid, sizeof(int) * launch_struct.size[0] * launch_struct.size[1], hipMemcpyDeviceToHost, stream);
    }

    // Free device memory
    hipFree(device_grid);
    // Free pinned host memory
    hipHostFree(grid);
    hipHostFree(magnetisation);
}

void launchModel2(hipStream_t stream, hiprandState *state, ising_model_config launch_struct) {
    // This model launches a kernel that is fully initilised on host working in shared memory. 
    // This function initilises the device memory and launches the kernel, then collects the result and frees the device memory.
    // The kernel is defined in kernels.h
    // The kernel is launched on the stream passed as argument.

    // Create pointers to device memory
    float *device_grid;
    // Allocate device memory
    hipMalloc((void **) &device_grid, sizeof(int) * launch_struct.size[0] * launch_struct.size[1] * launch_struct.num_concurrent);
    // Allocate pinned host memory
    int *grid;
    hipHostMalloc((void **) &grid, sizeof(int) * launch_struct.size[0] * launch_struct.size[1] * launch_struct.num_concurrent);
    int *magnetisation;
    hipHostMalloc((void **) &magnetisation, sizeof(int) * launch_struct.num_concurrent);

    for(int i=0; i<launch_struct.iterations; i+=launch_struct.iter_per_step) {
        // Launch kernel
        // ising_kernel_many<<<launch_struct.num_concurrent, 1, 0, stream>>>(state, device_grid, launch_struct.size[0], launch_struct.size[1], launch_struct.iter_per_step, launch_struct.temperature);
        // compute_magnetisation<<<launch_struct.num_concurrent, 1, 0, stream>>>(device_grid, launch_struct.size[0], launch_struct.size[1], magnetisation);
        
        // Collect result
        hipMemcpyAsync(grid, device_grid, sizeof(int) * launch_struct.size[0] * launch_struct.size[1], hipMemcpyDeviceToHost, stream);
    }

    // Free device memory
    hipFree(device_grid);
    // Free pinned host memory
    hipHostFree(grid);
    hipHostFree(magnetisation);
}

void launchModel3(hipStream_t stream, hiprandState *state, ising_model_config launch_struct) {
    // This model launches a kernel that is fully initilised on host working in shared memory. 
    // This function initilises the device memory and launches the kernel, then collects the result and frees the device memory.
    // The kernel is defined in kernels.h
    // The kernel is launched on the stream passed as argument.

    // Create pointers to device memory
    float *device_grid;
    // Allocate device memory
    hipMalloc((void **) &device_grid, sizeof(int) * launch_struct.size[0] * launch_struct.size[1] * launch_struct.num_concurrent);
    // Allocate pinned host memory
    int *grid;
    hipHostMalloc((void **) &grid, sizeof(int) * launch_struct.size[0] * launch_struct.size[1] * launch_struct.num_concurrent);
    int *magnetisation;
    hipHostMalloc((void **) &magnetisation, sizeof(int) * launch_struct.num_concurrent);

    for(int i=0; i<launch_struct.iterations; i+=launch_struct.iter_per_step) {
        // Launch kernel
        // ising_kernel_many<<<launch_struct.num_concurrent, 1, 0, stream>>>(state, device_grid, launch_struct.size[0], launch_struct.size[1], launch_struct.iter_per_step, launch_struct.temperature);
        // compute_magnetisation<<<launch_struct.num_concurrent, 1, 0, stream>>>(device_grid, launch_struct.size[0], launch_struct.size[1], magnetisation);
        
        // Collect result
        hipMemcpyAsync(grid, device_grid, sizeof(int) * launch_struct.size[0] * launch_struct.size[1], hipMemcpyDeviceToHost, stream);
    }

    // Free device memory
    hipFree(device_grid);
    // Free pinned host memory
    hipHostFree(grid);
    hipHostFree(magnetisation);
}