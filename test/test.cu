#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#include "../include/input_reader.h"

int main( int argc, char *argv[]) {
    char* filename = "input.txt";
    // Check if command line arguments were provided
    if (argc > 1) {
        filename = argv[1];
    }
    // pattern for further arguments
    // if (argc > 2) {
    //     count = atoi(argv[2]);
    // } 
    if (argc > 2) {
        printf("Usage: %s [filename] \n", argv[0]);
        exit(1);
    }
    int models;
    get_number_of_models(filename, &models);
    // Allocate memory for the array of pointers to structs
    ising_model_config* params_array[models];
    if (params_array == NULL) {
        fprintf(stderr, "Error: Could not allocate memory\n");
        exit(1);
    }
    read_input_file(filename, params_array, models);
    fprintf(stderr, "Number of models: %d\n", sizeof(params_array)/sizeof(params_array[0]));
    
    // Rest of the program...
    
    return 0;
    }